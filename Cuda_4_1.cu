#include "hip/hip_runtime.h"
#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cub/device/device_reduce.cuh>
#include <cub/block/block_reduce.cuh>

#define NUM_OF_BLOCKS   128


__global__ void makeGrid(double* outArray, int arraySize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < arraySize && j < arraySize)
            outArray[i * arraySize + j] = 0;
}

__global__ void setBorders(double* outArray, int arraySize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < arraySize)
    {
        outArray[i] = 10.0 + 10.0 * i / (arraySize - 1);
        outArray[i * arraySize] = 10.0 + 10.0 * i / (arraySize - 1);
        outArray[i * arraySize + arraySize - 1] = 20.0 + 10.0 * i / (arraySize - 1);
        outArray[arraySize * (arraySize - 1)+ i] = 20.0 + 10.0 * i / (arraySize - 1);
    }
}

__global__ void calcMatrix(double* Array1, double* Array2, int arraySize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i > 0 && i < arraySize-1 && j > 0 && j < arraySize-1)
        Array1[i * arraySize + j] = (Array2[(j + 1) * arraySize + (i + 1)]
                                    + Array2[(j + 1) * arraySize + (i - 1)]
                                    + Array2[(j - 1) * arraySize + (i + 1)]
                                    + Array2[(j - 1) * arraySize + (i - 1)]) / 4;
}

__global__ void matrixDiff(double* Array1, double* Array2, int arraySize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    Array1[i * arraySize + j] = fabs(Array2[i * arraySize + j] - Array1[i * arraySize + j]);
}

template <int BLOCK_THREADS>
__global__ void MyReduce(double *d_in, double* d_out, int arraySize)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    size_t j = blockIdx.y * blockDim.y + threadIdx.y;
    typedef hipcub::BlockReduce<double, BLOCK_THREADS> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    double data = 0;
    data = d_in[i * arraySize + j];
    double aggregate = BlockReduce(temp_storage).Reduce(data, hipcub::Max());
    if (threadIdx.x == 0)
        d_out[blockIdx.x] = aggregate;

}


int main(int argc, char* argv[])
{   
    double error;
    int size, iterations, c1, c2, c3;
    if (argc < 4)
    {
        error = 0.01;
        size = 128;
        iterations = 100;
    }
    else if (argc == 4)
    {
        c1 = sscanf(argv[1], "%lf", &error);
        c2 = sscanf(argv[2], "%i", &size);
        c3 = sscanf(argv[3], "%i", &iterations);
        if (c1 != 1 || c2 != 1 || c3 != 1)
        {
            fprintf(stderr, "Error: invalid command line argument. \n");
            return EXIT_FAILURE;
        }
    }
    else 
    {
        fprintf(stderr, "Error: invalid number of arguments. \n");
        return EXIT_FAILURE;
    }

    // set matrix 
    double* dA1;
    double* dA2;
    double* A1 = (double*) malloc(size * size * sizeof(double));
    double* A2 = (double*) malloc(size * size * sizeof(double));

    hipMalloc(&dA1, size * size * sizeof(double));
    hipMalloc(&dA2, size * size * sizeof(double));
    
    int gridParam = 1;
    if (size % 16 == 0)
        gridParam = 16;
    int blockParam = size / gridParam;

    dim3 BS(size, 1);
	dim3 GS(ceil(size/(float)BS.x), ceil(size/(float)BS.y));

    makeGrid<<<BS, GS>>>(dA1, size);
    makeGrid<<<BS, GS>>>(dA2, size);
    setBorders<<<blockParam, gridParam>>>(dA1, size);
    setBorders<<<blockParam, gridParam>>>(dA2, size);

    // set error variable
    double currentError = 1;
    double *d_currentError;
    hipMalloc(&d_currentError, sizeof(double));

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, dA1, d_currentError, size*size);
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    hipStream_t stream;
    hipStreamCreate(&stream);
    bool graphCreated = false;
    hipGraph_t graph;
    hipGraphExec_t instance;

    double* cub_result;
    hipMalloc(&cub_result, NUM_OF_BLOCKS * sizeof(double));

    // calculations
    clock_t start = clock();
    int k = 0;
    while (k < iterations & currentError > error)
    {
        
        if(!graphCreated)
        {
            hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
            calcMatrix<<<BS, GS, 0, stream>>>(dA1, dA2, size);
            calcMatrix<<<BS, GS, 0, stream>>>(dA2, dA1, size);
            hipStreamEndCapture(stream, &graph);
            hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
            graphCreated=true;
        }

        hipGraphLaunch(instance, stream);
        hipStreamSynchronize(stream);

        if (k % 100 == 0)
        {
            hipStreamSynchronize(stream);
            matrixDiff<<<BS, GS>>>(dA1, dA2, size);
            MyReduce<NUM_OF_BLOCKS><<<BS, GS>>>(dA1, cub_result, NUM_OF_BLOCKS);
            hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, cub_result, d_currentError, NUM_OF_BLOCKS);
            hipMemcpy(&currentError, d_currentError, sizeof(double), hipMemcpyDeviceToHost);
            setBorders<<<blockParam, gridParam>>>(dA1, size);
        }
        k += 2;
    }
    clock_t end = clock();

    printf("Error: %lf\n", currentError);
    printf("Number of iterations: %i\n", k);
    printf("Time: %lf\n", (double)(end - start) / CLOCKS_PER_SEC);

    hipStreamDestroy(stream);
    free(A1); free(A2);
    hipFree(dA1); hipFree(dA2);
    return 0;
}
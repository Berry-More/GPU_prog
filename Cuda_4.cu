#include "hip/hip_runtime.h"
#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cub/device/device_reduce.cuh>


__global__ void makeGrid(double* outArray, int arraySize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < arraySize && j < arraySize)
            outArray[i * arraySize + j] = 0;
}

__global__ void setBorders(double* outArray, int arraySize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < arraySize)
    {
        outArray[i] = 10.0 + 10.0 * i / (arraySize - 1);
        outArray[i * arraySize] = 10.0 + 10.0 * i / (arraySize - 1);
        outArray[i * arraySize + arraySize - 1] = 20.0 + 10.0 * i / (arraySize - 1);
        outArray[arraySize * (arraySize - 1)+ i] = 20.0 + 10.0 * i / (arraySize - 1);
    }
}

__global__ void calcMatrix(double* Array1, double* Array2, int arraySize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i > 0 && i < arraySize-1 && j > 0 && j < arraySize-1)
        Array1[i * arraySize + j] = (Array2[(i - 1) * arraySize + (j + 1)]
                                    + Array2[(i - 1) * arraySize + (j - 1)]
                                    + Array2[(i + 1) * arraySize + (j + 1)]
                                    + Array2[(i + 1) * arraySize + (j - 1)]) / 4;
}

__global__ void matrixDiff(double* Array1, double* Array2, int arraySize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    Array1[i * arraySize + j] = fabs(Array2[i * arraySize + j] - Array1[i * arraySize + j]);
}


int main(int argc, char* argv[])
{   
    double error;
    int size, iterations, c1, c2, c3;
    if (argc < 4)
    {
        error = 0.01;
        size = 128;
        iterations = 100;
    }
    else if (argc == 4)
    {
        c1 = sscanf(argv[1], "%lf", &error);
        c2 = sscanf(argv[2], "%i", &size);
        c3 = sscanf(argv[3], "%i", &iterations);
        if (c1 != 1 || c2 != 1 || c3 != 1)
        {
            fprintf(stderr, "Error: invalid command line argument. \n");
            return EXIT_FAILURE;
        }
    }
    else 
    {
        fprintf(stderr, "Error: invalid number of arguments. \n");
        return EXIT_FAILURE;
    }

    // set matrix 
    double* dA1;
    double* dA2;
    double* A1 = (double*) malloc(size * size * sizeof(double));
    double* A2 = (double*) malloc(size * size * sizeof(double));

    hipMalloc(&dA1, size * size * sizeof(double));
    hipMalloc(&dA2, size * size * sizeof(double));
    
    int gridParam = 1;
    if (size % 16 == 0)
        gridParam = 16;
    int blockParam = size / gridParam;

    dim3 threadsPerBlock(gridParam, gridParam);
    dim3 numBlocks(blockParam, blockParam);

    makeGrid<<<numBlocks, threadsPerBlock>>>(dA1, size);
    makeGrid<<<numBlocks, threadsPerBlock>>>(dA2, size);
    setBorders<<<blockParam, gridParam>>>(dA1, size);
    setBorders<<<blockParam, gridParam>>>(dA2, size);

    // set error variable
    double currentError = 1;
    double *d_currentError;
    hipMalloc(&d_currentError, sizeof(double));

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, dA1, d_currentError, size*size);
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // calculations
    clock_t start = clock();
    int k = 0;
    while (k < iterations & currentError > error)
    {
        calcMatrix<<<numBlocks, threadsPerBlock>>>(dA1, dA2, size);
        calcMatrix<<<numBlocks, threadsPerBlock>>>(dA2, dA1, size);

        if (k % 100 == 0)
        {
            matrixDiff<<<numBlocks, threadsPerBlock>>>(dA1, dA2, size);
            hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, dA1, d_currentError, size*size);
            hipMemcpy(&currentError, d_currentError, sizeof(double), hipMemcpyDeviceToHost);
            setBorders<<<blockParam, gridParam>>>(dA1, size);
        }
        k += 2;
    }
    clock_t end = clock();

    printf("Error: %lf\n", currentError);
    printf("Number of iterations: %i\n", k);
    printf("Time: %lf\n", (double)(end - start) / CLOCKS_PER_SEC);

    free(A1); free(A2);
    hipFree(dA1); hipFree(dA2);
    return 0;
}